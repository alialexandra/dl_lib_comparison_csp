#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#ifndef N
#define N 1024
#endif

#ifndef TILE_SIZE
#define TILE_SIZE 16
#endif

#ifndef NUM_REPS
#define NUM_REPS 3
#endif

__global__ void matrixMulShared(const double *A, const double *B, double *C, int n)
{
    __shared__ double tileA[TILE_SIZE][TILE_SIZE];
    __shared__ double tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    double sum = 0.0;

    for (int tile = 0; tile < n / TILE_SIZE; ++tile)
    {
        tileA[threadIdx.y][threadIdx.x] = A[row * n + tile * TILE_SIZE + threadIdx.x];
        tileB[threadIdx.y][threadIdx.x] = B[(tile * TILE_SIZE + threadIdx.y) * n + col];
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        __syncthreads();
    }

    C[row * n + col] = sum;
}

double gpu_timer(hipEvent_t start, hipEvent_t stop)
{
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    return ms / 1000.0;
}

int main()
{
    int size = N * N * sizeof(double);
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks(N / TILE_SIZE, N / TILE_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipEventRecord(start);
        matrixMulShared<<<blocks, threads>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        total += gpu_timer(start, stop);
    }

    printf("GPU Shared Memory: N=%d TILE_SIZE=%d → Avg time = %.6f seconds\n",
           N, TILE_SIZE, total / NUM_REPS);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

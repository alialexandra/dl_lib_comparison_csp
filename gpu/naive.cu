#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef NUM_REPS
#define NUM_REPS 3
#endif

__global__ void matrixMulKernel(const double *A, const double *B, double *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n)
    {
        double sum = 0.0;
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}

double gpu_timer(hipEvent_t start, hipEvent_t stop)
{
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    return ms / 1000.0; // seconds
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    if (N <= 0)
    {
        fprintf(stderr, "Invalid matrix size: %d\n", N);
        return 1;
    }

    printf("Running matrix multiplication with N = %d\n", N);

    int size = N * N * sizeof(double);
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipEventRecord(start);
        matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        total += gpu_timer(start, stop);
    }

    // Log memory info
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);

    // Print summary
    double avg_time = total / NUM_REPS;
    printf("Naive GPU: N=%d → Avg time = %.6f seconds\n", N, avg_time);

    // Save results to CSV
    FILE *log = fopen("naive_gpu_results.csv", "a");
    if (log)
    {
        fprintf(log, "%d,%d,%d,%.6f,%zu,%zu\n",
                N, threadsPerBlock.x, blocksPerGrid.x,
                avg_time, total_mem, free_mem);
        fclose(log);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

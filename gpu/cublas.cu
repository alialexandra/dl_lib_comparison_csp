#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef N
#define N 1024
#endif

#ifndef NUM_REPS
#define NUM_REPS 3
#endif

int main()
{
    int size = N * N * sizeof(double);
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, beta = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipEventRecord(start);

        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total += ms / 1000.0;
    }

    printf("cuBLAS DGEMM: N=%d → Avg time = %.6f seconds\n", N, total / NUM_REPS);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

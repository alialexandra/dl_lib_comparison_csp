#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef NUM_REPS
#define NUM_REPS 5
#endif

double gpu_timer(hipEvent_t start, hipEvent_t stop)
{
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    return ms / 1000.0;
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    size_t size = N * N * sizeof(double);

    // Host memory
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size); // final result

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
        h_C[i] = 0.0;
    }

    // Device memory
    double *d_A, *d_B, *d_C, *d_temp;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);    // will hold B·Aᵀ + A²·B
    hipMalloc(&d_temp, size); // for intermediate matrices

    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "Memcpy A");
    checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "Memcpy B");

    // cuBLAS setup
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const double alpha = 1.0, beta = 0.0;

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total_time = 0.0;

    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipMemset(d_temp, 0, size);

        hipEventRecord(start);

        // 1. d_C = B × Aᵀ
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                    N, N, N, &alpha,
                    d_B, N, d_A, N, &beta,
                    d_C, N);

        // 2. d_temp = A × A
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N, &alpha,
                    d_A, N, d_A, N, &beta,
                    d_temp, N);

        // 3. d_temp = A² × B
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N, &alpha,
                    d_temp, N, d_B, N, &beta,
                    d_temp, N);

        // 4. Final sum: d_C += d_temp
        hipblasDaxpy(handle, N * N, &alpha, d_temp, 1, d_C, 1);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        total_time += gpu_timer(start, stop);
    }

    double total = total_time / NUM_REPS;
    double gflops = (6.0 * N * N * N) / (total * 1e9);
    printf("%.6f %.2f\n", total, gflops);

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_temp);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
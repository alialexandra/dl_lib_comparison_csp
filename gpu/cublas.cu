#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef NUM_REPS
#define NUM_REPS 3
#endif

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    size_t size = N * N * sizeof(double);

    // Host memory allocation
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);
    if (!h_A || !h_B || !h_C)
    {
        fprintf(stderr, "Host memory allocation failed\n");
        return 1;
    }

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    // Device memory allocation
    double *d_A, *d_B, *d_C;
    if (hipMalloc(&d_A, size) != hipSuccess ||
        hipMalloc(&d_B, size) != hipSuccess ||
        hipMalloc(&d_C, size) != hipSuccess)
    {
        fprintf(stderr, "Device memory allocation failed (likely out of memory)\n");
        free(h_A);
        free(h_B);
        free(h_C);
        return 1;
    }

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, beta = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipEventRecord(start);
        hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                          N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
            fprintf(stderr, "cuBLAS DGEMM failed for N = %d\n", N);
            break;
        }

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total += ms / 1000.0;
    }

    double avg_time = total / NUM_REPS;
    printf("cuBLAS DGEMM: N=%d → Avg time = %.6f seconds\n", N, avg_time);

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

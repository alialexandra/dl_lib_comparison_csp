#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int NUM_REPS = 3;

    size_t size = N * N * sizeof(double);
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, beta = 0.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double total = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipEventRecord(start);
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total += ms / 1000.0;
    }

    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);

    double avg_time = total / NUM_REPS;
    printf("cuBLAS DGEMM: N=%d → Avg time = %.6f seconds\n", N, avg_time);

    FILE *log = fopen("cublas_results.csv", "a");
    if (log)
    {
        fprintf(log, "%d,%.6f,%zu,%zu\n", N, avg_time, total_mem, free_mem);
        fclose(log);
    }

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}

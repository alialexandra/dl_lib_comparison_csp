#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef NUM_REPS
#define NUM_REPS 10
#endif

void checkCublas(hipblasStatus_t stat, const char *msg)
{
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS error: %s (code %d)\n", msg, stat);
        exit(EXIT_FAILURE);
    }
}

void checkCuda(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s (%s)\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
        fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

    hipFree(0);

    int N = atoi(argv[1]);
    size_t size = N * N * sizeof(double);

    // Host memory
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);
    if (!h_A || !h_B || !h_C)
    {
        fprintf(stderr, "Host malloc failed\n");
        return 1;
    }

    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    // Device memory
    double *d_A, *d_B, *d_C;
    checkCuda(hipMalloc(&d_A, size), "hipMalloc d_A");
    checkCuda(hipMalloc(&d_B, size), "hipMalloc d_B");
    checkCuda(hipMalloc(&d_C, size), "hipMalloc d_C");

    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "Memcpy A");
    checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "Memcpy B");

    // cuBLAS setup
    hipblasHandle_t handle;
    checkCublas(hipblasCreate(&handle), "create handle");

    double alpha = 1.0, beta = 0.0;

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float total_ms = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
        hipMemset(d_C, 0, size);
        hipEventRecord(start);

        checkCublas(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N),
                    "hipblasDgemm");

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_ms += ms;
    }

    float avg_time_ms = total_ms / NUM_REPS;
    printf("cuBLAS: N=%d → Avg time = %.6f ms\n", N, avg_time_ms);

    // Copy result back
    checkCuda(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost), "Memcpy C");

    // Print example values
    printf("some of the results: C[0] = %f, C[%d] = %f\n", h_C[0], N * N - 1, h_C[N * N - 1]);

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

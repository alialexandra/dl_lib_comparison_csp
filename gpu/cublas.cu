#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifndef NUM_REPS
<<<<<<< Updated upstream
#define NUM_REPS 10
#endif

void checkCublas(hipblasStatus_t stat, const char *msg)
{
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuBLAS error: %s (code %d)\n", msg, stat);
        exit(EXIT_FAILURE);
    }
}

void checkCuda(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s (%s)\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv)
{
    if (argc < 2)
    {
=======
#define NUM_REPS 5
#endif

double gpu_timer(hipEvent_t start, hipEvent_t stop) {
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    return ms / 1000.0;
}

int main(int argc, char **argv) {
    if (argc < 2) {
>>>>>>> Stashed changes
        fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
        return 1;
    }

<<<<<<< Updated upstream
    hipFree(0);

=======
>>>>>>> Stashed changes
    int N = atoi(argv[1]);
    size_t size = N * N * sizeof(double);

    // Host memory
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
<<<<<<< Updated upstream
    double *h_C = (double *)malloc(size);
    if (!h_A || !h_B || !h_C)
    {
        fprintf(stderr, "Host malloc failed\n");
        return 1;
    }
=======
    double *h_C = (double *)malloc(size);  // final result
>>>>>>> Stashed changes

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
        h_C[i] = 0.0;
    }

    // Device memory
<<<<<<< Updated upstream
    double *d_A, *d_B, *d_C;
    checkCuda(hipMalloc(&d_A, size), "hipMalloc d_A");
    checkCuda(hipMalloc(&d_B, size), "hipMalloc d_B");
    checkCuda(hipMalloc(&d_C, size), "hipMalloc d_C");
=======
    double *d_A, *d_B, *d_C, *d_temp;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);     // will hold B·Aᵀ + A²·B
    hipMalloc(&d_temp, size);  // for intermediate matrices
>>>>>>> Stashed changes

    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "Memcpy A");
    checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "Memcpy B");

    // cuBLAS setup
    hipblasHandle_t handle;
<<<<<<< Updated upstream
    checkCublas(hipblasCreate(&handle), "create handle");

    double alpha = 1.0, beta = 0.0;
=======
    hipblasCreate(&handle);
    const double alpha = 1.0, beta = 0.0;
>>>>>>> Stashed changes

    // Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

<<<<<<< Updated upstream
    float total_ms = 0.0;
    for (int rep = 0; rep < NUM_REPS; ++rep)
    {
=======
    double total_time = 0.0;

    for (int rep = 0; rep < NUM_REPS; ++rep) {
>>>>>>> Stashed changes
        hipMemset(d_C, 0, size);
        hipMemset(d_temp, 0, size);

        hipEventRecord(start);

<<<<<<< Updated upstream
        checkCublas(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N),
                    "hipblasDgemm");

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        total_ms += ms;
    }

    float avg_time_ms = total_ms / NUM_REPS;
    printf("cuBLAS: N=%d → Avg time = %.6f ms\n", N, avg_time_ms);

    // Print example values
    printf("some of the results: C[0] = %f, C[%d] = %f\n", d_C[0], N * N - 1, d_C[N * N - 1]);

    // Clean up
=======
        // 1. d_C = B × Aᵀ
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                    N, N, N, &alpha,
                    d_B, N, d_A, N, &beta,
                    d_C, N);

        // 2. d_temp = A × A
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N, &alpha,
                    d_A, N, d_A, N, &beta,
                    d_temp, N);

        // 3. d_temp = A² × B
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    N, N, N, &alpha,
                    d_temp, N, d_B, N, &beta,
                    d_temp, N);

        // 4. Final sum: d_C += d_temp
        hipblasDaxpy(handle, N * N, &alpha, d_temp, 1, d_C, 1);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        total_time += gpu_timer(start, stop);
    }

    double total = total_time / NUM_REPS;
    double gflops = (6.0 * N * N * N) / (total * 1e9);
    printf("%.2f %.2f\n", total, gflops);

    // Cleanup
>>>>>>> Stashed changes
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_temp);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}